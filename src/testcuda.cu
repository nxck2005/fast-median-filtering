#include <hip/hip_runtime.h>
#include <stdio.h> // For fprintf

// --- Add this error checking macro ---
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
// ------------------------------------

__global__ void addKernel(int *c, const int *a, const int *b) {
    *c = *a + *b;
}

int main() {
    int a = 10;
    int b = 25;
    int c = 0;

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // --- Wrap all CUDA calls in the macro ---
    gpuErrchk(hipMalloc((void**)&dev_a, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_b, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_c, sizeof(int)));

    gpuErrchk(hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice));

    addKernel<<<1, 1>>>(dev_c, dev_a, dev_b);
    
    // Check for errors from the asynchronous kernel launch
    gpuErrchk(hipGetLastError());

    // Synchronize and check for any lingering errors
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

    printf("Hello from the GPU! The result is %d\n", c);

    gpuErrchk(hipFree(dev_a));
    gpuErrchk(hipFree(dev_b));
    gpuErrchk(hipFree(dev_c));
    
    return 0;
}